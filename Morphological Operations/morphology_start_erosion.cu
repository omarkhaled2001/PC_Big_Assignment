#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib> // for atoi

using namespace cv;

#define CHECK_CUDA_ERROR(call)                                           \
    do                                                                   \
    {                                                                    \
        hipError_t err = call;                                          \
        if (err != hipSuccess)                                          \
        {                                                                \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err));        \
            exit(EXIT_FAILURE);                                          \
        }                                                                \
    } while (0)

// Kernel to perform erosion
__global__ void erosionKernel(unsigned char *input, unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int pixel_value = 255; // Assume a binary image where 255 is white and 0 is black
        for (int ky = -1; ky <= 1; ++ky)
        {
            for (int kx = -1; kx <= 1; ++kx)
            {
                int ix = x + kx;
                int iy = y + ky;
                if (ix >= 0 && ix < width && iy >= 0 && iy < height)
                {
                    pixel_value = min(pixel_value, input[iy * width + ix]);
                }
            }
        }
        output[y * width + x] = pixel_value;
    }
}

// Kernel to perform dilation
__global__ void dilationKernel(unsigned char *input, unsigned char *output, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int pixel_value = 0; // Assume a binary image where 255 is white and 0 is black
        for (int ky = -1; ky <= 1; ++ky)
        {
            for (int kx = -1; kx <= 1; ++kx)
            {
                int ix = x + kx;
                int iy = y + ky;
                if (ix >= 0 && ix < width && iy >= 0 && iy < height)
                {
                    pixel_value = max(pixel_value, input[iy * width + ix]);
                }
            }
        }
        output[y * width + x] = pixel_value;
    }
}

int main(int argc, char **argv)
{
    if (argc < 4)
    {
        fprintf(stderr, "Usage: %s <image_path>, %d <erosion_iterations> , %d <dilation_iterations>  \n", argv[0], argv[1], argv[2]);
        return -1;
    }

    // Load image using OpenCV
    Mat image = imread(argv[1], IMREAD_GRAYSCALE);
    if (image.empty())
    {
        fprintf(stderr, "Could not open or find the image\n");
        return -1;
    }

    int width = image.cols;
    int height = image.rows;

    // Number of iterations for erosion
    int erosion_iterations = atoi(argv[2]);

    // Number of iterations for dilation
    int dilation_iterations = atoi(argv[3]);

    // Allocate memory for the input and output images on the device
    unsigned char *d_input;
    unsigned char *d_output;
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_input, width * height * sizeof(unsigned char)));
    CHECK_CUDA_ERROR(hipMalloc((void **)&d_output, width * height * sizeof(unsigned char)));

    // Copy the input image to device memory
    CHECK_CUDA_ERROR(hipMemcpy(d_input, image.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Define block and grid dimensions
    dim3 block(16, 16);
    dim3 grid((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);

    // Perform multiple iterations of the erosion kernel
    for (int i = 0; i < erosion_iterations; ++i)
    {
        erosionKernel<<<grid, block>>>(d_input, d_output, width, height);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        CHECK_CUDA_ERROR(hipMemcpy(d_input, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice));
    }

    // Copy the result back to host and save it
    CHECK_CUDA_ERROR(hipMemcpy(image.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
    imwrite("eroded_image.png", image);

    // Reset the input image on device memory for dilation
    CHECK_CUDA_ERROR(hipMemcpy(d_input, image.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice));

    // Perform dilation kernel once for demonstration
    for (int i = 0; i < dilation_iterations; ++i)
    {
        dilationKernel<<<grid, block>>>(d_input, d_output, width, height);
        CHECK_CUDA_ERROR(hipGetLastError());
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        CHECK_CUDA_ERROR(hipMemcpy(d_input, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToDevice));
    }

    // Copy the result back to host and save it
    CHECK_CUDA_ERROR(hipMemcpy(image.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost));
    imwrite("dilated_image.png", image);

    // Free device memory
    CHECK_CUDA_ERROR(hipFree(d_input));
    CHECK_CUDA_ERROR(hipFree(d_output));

    return 0;
}
